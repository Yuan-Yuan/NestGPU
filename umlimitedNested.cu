#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/fcntl.h>
#include <sys/mman.h>
#include <sys/socket.h>
#include <netdb.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <algorithm> 
#include <hipcub/hipcub.hpp>
#include <cub/grid/grid_barrier.cuh>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>

#define MAX_INT 1024*1024*1024
#define BILLION             1000000000

#define CUDACHECK(cmd) do { \
    hipError_t e = cmd; \
    pid_t pid = getpid();\
    if( e != hipSuccess ) { \
    printf("Porcess %d Failed: Cuda error %s:%d '%s'\n", \
    pid,__FILE__,__LINE__,hipGetErrorString(e)); \
    exit(EXIT_FAILURE); \
    } \
} while(0) 

#define NP2(n)              do {                    \
n--;                                            \
n |= n >> 1;                                    \
n |= n >> 2;                                    \
n |= n >> 4;                                    \
n |= n >> 8;                                    \
n |= n >> 16;                                   \
n ++; } while (0) 

#define TABLE2 0x8000000000000000
template<typename vec_t>
__global__ static void find_MSB(vec_t* data,int * msb,int size)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;
    vec_t tmp,t_data;
    int l_msb = 0;

    __shared__ int s_msb;
    __shared__ unsigned	long long max_data;

    s_msb = 0;
    max_data = 0;

    __syncthreads();

    for (int i = offset; i<size; i += stride){
    	t_data = data[offset];
    	if(max_data < t_data)
    	{
	    	for(int s = 0; s < 64; s++)
	    	{
		    	tmp = t_data >> s;
		        if(tmp != 0) l_msb = s;
		        else break;
		    }
		    atomicMax(&s_msb,l_msb);
		    atomicMax(&max_data,t_data);
		}
    }

    __syncthreads();

    if(threadIdx.x == 0)
    	atomicMax(&*msb,s_msb);
}

template<typename vec_t>
__global__ static void find_MAX(vec_t* data,vec_t * msb,int size)
{
    int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;
    vec_t t_data;

    __shared__ vec_t max_data;

    max_data = 0;

    __syncthreads();

    for (int i = offset; i<size; i += stride){
        t_data = data[offset];
        if(max_data < t_data)
            atomicMax(&max_data,t_data);
    }

    __syncthreads();

    if(threadIdx.x == 0)
        atomicMax(&*msb,max_data);
}

template<typename vec_t>
__global__ static void assign_index(vec_t *dim, long  inNum){
    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = offset; i<inNum; i += stride)
        dim[i] = i;
}

template<typename vec_t>
int get_partition(vec_t * data,int size,vec_t min,vec_t max,int * hist,int*psum,int hist_size,int myrank)
{
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0, temp_storage_bytes1 = 0;
    hipcub::DeviceHistogram::HistogramEven(d_temp_storage, temp_storage_bytes,
        data, hist, hist_size+1, min, max, size);

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes1, hist, psum, hist_size);
    assert(temp_storage_bytes1 < temp_storage_bytes);
// Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
// Compute histograms
    hipcub::DeviceHistogram::HistogramEven(d_temp_storage, temp_storage_bytes,
        data, hist, hist_size+1, min, max, size);
    // hipFree(d_temp_storage);
    hipMemset(d_temp_storage, 0, temp_storage_bytes1);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes1, hist, psum, hist_size);
    hipFree(d_temp_storage);
//printf("device %d CUB psum needs %d aux memory\n",myrank,temp_storage_bytes);
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "device %d get partition failed!\n",myrank);
        return 0;
    }
    return 1;
}

#define MASK 0x000000FF
template<typename vec_t>
__global__ static void probe_join_result(vec_t* gpu_dim, vec_t * gpu_dim_index, int* dim_psum, int dim_size, vec_t* gpu_fact,
 vec_t * gpu_fact_index, int* fact_psum,int fact_size, int hsize, vec_t * factFilter,int *pool_ptr,int *gpu_count){

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int stride = blockDim.x;
    int block_stride = gridDim.x;

    vec_t dim_data;
    vec_t dim_index;
    vec_t tmp_key[5];
    vec_t tmp_index[5];

    __shared__ vec_t fact_data[300];
    __shared__ vec_t fact_data_index[300];
    __shared__ int HT[256];

    typedef hipcub::BlockReduce<int, 256> BlockReduce;
    typedef hipcub::BlockScan<int, 256> BlockScan;

    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ typename BlockScan::TempStorage temp_storage1;

    int inner = 0;
    int fact_num;
    int count = 0;
    int thread_data;
    int block_aggregate;

    for (int i = bid; i<hsize; i += block_stride){  

        for(int j = tid; j < 300; j += stride)
        {
            fact_data[j] = MAX_INT;
            if(j < 256)
                HT[j] = 0;
        }
        
        __syncthreads();
        int dim_head = dim_psum[i];
        int dim_tail = i == hsize -1 ? dim_size : dim_psum[i+1];
        int fact_head = fact_psum[i];
        int fact_tail = i == hsize -1 ? fact_size : fact_psum[i+1];

        fact_num = fact_tail - fact_head > 300 ? 300 : fact_tail - fact_head;
        inner = 0;
        for(int j = tid + fact_head; j< fact_num + fact_head; j+= stride)
        {
            assert(j < fact_size);
            tmp_key[inner] = gpu_fact[j];
            tmp_index[inner] = gpu_fact_index[j];
            int key = tmp_key[inner] & MASK ;
            atomicAdd(&HT[key],1);
            inner ++;
        }
        __syncthreads();

        thread_data = HT[threadIdx.x];

        __syncthreads();

        BlockScan(temp_storage1).ExclusiveSum(thread_data, thread_data, block_aggregate);

        __syncthreads();

        HT[threadIdx.x] = thread_data;

        __syncthreads();

        for(int j = 0; j < inner; j++)
        {
            int key = tmp_key[j] & MASK ;
            int pos = atomicAdd(&HT[key],1);
            fact_data[pos] = tmp_key[j];
            fact_data_index[pos] = tmp_index[j];
        }

        __syncthreads();

        inner = 0;
        for(int j = tid + dim_head; j< dim_tail; j+= stride)
        {
            assert(j < dim_size);
            dim_data = gpu_dim[j];
            dim_index = gpu_dim_index[j];
            int key = dim_data & MASK;
            int search_num = key == 0? HT[0] : HT[key] - HT[key - 1];
            int pos = key == 0? 0 : HT[key - 1];
            for(int z = 0; z < search_num; z++)
                if(fact_data[pos + z] == dim_data)
                    count += 3;
        }
        __syncthreads();      
    }

    int aggregate = BlockReduce(temp_storage).Sum(count);
    __syncthreads();
    
    if(threadIdx.x == 0)
        atomicAdd(&(*pool_ptr), aggregate); 
}

__device__ int binarySearchLowerBound(int* A, int target, int n){
    int low = 0, high = n, mid;
    while(low <= high){
        mid = low + (high - low) / 2;
        if(target <= A[mid]){
            high = mid - 1;
        }else{
            low = mid + 1;
        }
    }
    if(low < n && A[low] == target)
        return low;
    else
        return -1;
}

template<typename vec_t>
__global__ static void merge_join_result(vec_t* gpu_dim, vec_t * gpu_dim_index, vec_t * bucket, int* dim_psum, int dim_size, vec_t* gpu_fact, 
	vec_t * gpu_fact_index, int* fact_psum,int fact_size, int hsize, int * factFilter,int *gpu_count){

	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int stride = blockDim.x;
	int block_stride = gridDim.x;
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_num = stride * block_stride;
	__shared__ vec_t dim_data[2][512];
	// __shared__ vec_t dim_index[512];
	vec_t fact_data;
	vec_t fact_data_index;
	int inner = 0;
	int count = 0;
	int head,tail;

	for (int i = bid; i<hsize; i += block_stride){      
		for(int j=tid;j<512;j+=stride)
			dim_data[0][j] = MAX_INT;
		__syncthreads();

		int dim_head = dim_psum[i];
		int dim_tail = i == hsize -1 ? dim_size : dim_psum[i+1];
		int fact_head = fact_psum[i];
		int fact_tail = i == hsize -1 ? fact_size : fact_psum[i+1];
		assert(dim_tail - dim_head < 512);
		  //printf("bid %d tid %d head %d tail %d\n", i,tid,fact_head,fact_tail);
		inner = 0;
		for(int j = tid + dim_head; j< dim_tail; j+= stride)
		{
			dim_data[0][tid + inner] = bucket[j * 2];
			dim_data[1][tid + inner] = bucket[j * 2 + 1];
			inner += stride;
		}
		__syncthreads();

		for(int j = tid + fact_head; j< fact_tail; j+= stride)
		{
			fact_data = gpu_fact[j];
			fact_data_index = gpu_fact_index[j];

			// int re = binarySearchLowerBound(dim_data,fact_data,dim_tail - dim_head);
			// if(re != -1)
			// {
			// 	int fact_id = fact_data_index;
			// 	factFilter[fact_id] = dim_index[re];
			// 	count ++;
			// 	atomicAdd(&gpu_count[fact_id % thread_num],1);
			// }

			for(int k = 0; k < dim_tail - dim_head; k++)	
				if(fact_data == dim_data[0][k])
				{
					int fact_id = fact_data_index;
					factFilter[fact_id] = dim_data[1][k];
					count ++;
					atomicAdd(&gpu_count[fact_id % thread_num],1);
					break;
				}
		}
		__syncthreads();	
	}
	// gpu_count[offset] = count;
}


template<typename vec_t>
__global__ static void gpudb_like_probe(vec_t* gpu_dim, vec_t * gpu_dim_index, vec_t * bucket, int * num, int* dim_psum, int dim_size, vec_t* gpu_fact, 
	vec_t * gpu_fact_index, int fact_size, int hsize, int * factFilter,int *count){

	int lcount = 0;
    int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<fact_size;i+=stride){

        int fkey = ((int *)(gpu_fact))[i];
        int hkey = fkey &(hsize-1);
        int keyNum = num[hkey];
        int fvalue = 0;

        for(int j=0;j<keyNum;j++){
            int pSum = dim_psum[hkey];
            int dimKey = ((int *)(bucket))[2*j + 2*pSum];

            if(dimKey == fkey){

                int dimId = ((int *)(bucket))[2*j + 2*pSum + 1];
                lcount ++;
                fvalue = dimId;

                break;
            }
        }
        factFilter[i] = fvalue;
    }
    count[offset] = lcount;
}

template<typename vec_t>
__global__ static void merge_join_result_reuse(vec_t* gpu_dim, vec_t * gpu_dim_index, int* dim_psum, int dim_size, vec_t* gpu_fact, 
	vec_t * gpu_fact_index, int* fact_psum,int fact_size, int hsize, vec_t * factFilter,int *pool_ptr,vec_t *batch1,int num1,vec_t * batch2,uint num2,
	hipcub::GridBarrier global_barrier,int * flags, int * mutex, int * end,int buffer_size){

	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int stride = blockDim.x;
	int block_stride = gridDim.x;
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int reg_block = (hsize / block_stride + (hsize % block_stride == 0 ? 0 : 1)) * block_stride;
	vec_t dim_data[10];
	vec_t dim_index[10];
	__shared__ vec_t fact_data[400];
	__shared__ vec_t fact_data_index[400];
	int inner = 0;
	int count = 0;
	int pos ;
	
	for (int i = bid; i < reg_block; i += block_stride){   
		pos = 0; 
		if(i < hsize)  
		{
			for(int j=tid;j<400;j+=stride)
				fact_data[j] = MAX_INT;
			__syncthreads();

			int dim_head = dim_psum[i];
			int dim_tail = i == hsize -1 ? dim_size : dim_psum[i+1];
			int fact_head = fact_psum[i];
			int fact_tail = i == hsize -1 ? fact_size : fact_psum[i+1];
			if(fact_tail - fact_head > 400 && tid == 0)
				printf("bid %d tid %d head %d tail %d\n", i,tid,fact_head,fact_tail);
			inner = 0;
			for(int j = tid + fact_head; j< fact_tail; j+= stride)
			{
				fact_data[tid + inner] = gpu_fact[j];
				fact_data_index[tid + inner] = gpu_fact_index[j];
				inner += stride;
			}
			__syncthreads();
			inner = 0;
			for(int j = tid + dim_head; j< dim_tail; j+= stride)
			{
				dim_data[inner] = gpu_dim[j];
				dim_index[inner++] = gpu_dim_index[j];
			}

			int pre = 0; 
			for(int z=0;z<inner;z++)
			{
			    int high = 400;
			    int low = 0;
			    while(low <= high )
			    {
			    	count ++;
			        int mid = (high + low)/2;
			        assert(mid < 400);
			        if(fact_data[mid] < dim_data[z])
			            low = mid + 1;
			        else if(fact_data[mid] > dim_data[z])
			            high = mid - 1;
			        else
			        {
			            while(mid-1>=0 && fact_data[mid-1] == dim_data[z])
			            {mid--;}
			            pre = mid;
			            break;                  
			        }
			    }
			    for(int j = pre; j < 400; j++)  
			        if(fact_data[j] == dim_data[z]) 
			        {
			        	count ++;
						pos = atomicAdd(&(*pool_ptr), 3);
						if(pos < num1) batch1[pos] = dim_data[z]; else batch2[pos - num1] = dim_data[z];
						if(pos + 1 < num1) batch1[pos + 1] = fact_data_index[z]; else batch2[pos + 1 - num1] = fact_data_index[z];
						if(pos + 2 < num1) batch1[pos + 2] = dim_index[z]; else batch2[pos + 2 - num1] = dim_index[z];
						// factFilter[pos] = dim_data[z];
						// factFilter[pos+1] = fact_data_index[j];
						// factFilter[pos+2] = dim_index[z];
			        }
			        else
			            break;
			}
		}
		global_barrier.Sync();
		if(pos != 0 && (pos / 3) % buffer_size == 0 || i == hsize - 1)
		{
			// while(atomicCAS(&*mutex,0,1) != 0)
   //          {}
            atomicAdd(&*flags,1);
   //          int f = atomicExch(&*mutex,0);
			// assert(f != 0);
		}
		__syncthreads();
		if(threadIdx.x == 0 && i == hsize - 1)
			(*end) = 0;
	}
}

__global__ static void count_hash_num(int *dim, long  inNum,int *num,int hsize){
    
    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){
        int joinKey = ((int *)dim)[i];
        int hKey = joinKey & (hsize-1);
        atomicAdd(&(num[hKey]),1);
    }
}

__global__ static void see(int * data)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;
}

__global__ static void build_hash_table(int *dim, long inNum, int *psum, int * bucket,int hsize){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){
        int joinKey = ((int *) dim)[i]; 
        int hKey = joinKey & (hsize-1);
        int pos = atomicAdd(&psum[hKey],1) * 2;
        assert(pos < inNum * 2);
        ((int*)bucket)[pos] = joinKey;
        pos += 1;
        int dimId = i+1;
        ((int*)bucket)[pos] = dimId;
    }

}


__global__ static void count_join_result(int* num, int* psum, int* bucket, int* fact, long inNum, int* count, int * factFilter,int hsize){

    int lcount = 0;
    int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){

        int fkey = ((int *)(fact))[i];
        int hkey = fkey &(hsize-1);
        int keyNum = num[hkey];
        int fvalue = 0;

        for(int j=0;j<keyNum;j++){
            int pSum = psum[hkey];
            int dimKey = ((int *)(bucket))[2*j + 2*pSum];

            if(dimKey == fkey){

                int dimId = ((int *)(bucket))[2*j + 2*pSum + 1];
                lcount ++;
                fvalue = dimId;

                break;
            }
        }
        factFilter[i] = fvalue;
    }
    count[offset] = lcount;
}

__global__ static void index_join_result(int* num, int* psum, int* data, int * index, int* fact, long inNum, int* count, int * factFilter,int hsize){

    int lcount = 0;
    int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){

        int fkey = ((int *)(fact))[i];
        int hkey = fkey &(hsize-1);
        int keyNum = num[hkey];
        int fvalue = 0;

        for(int j=0;j<keyNum;j++){
            int pSum = psum[hkey];
            int dimKey = ((int *)(data))[j + pSum];

            if(dimKey == fkey){

                int dimId = ((int *)(index))[j + pSum];
                lcount ++;
                fvalue = dimId + 1;

                break;
            }
        }
        factFilter[i] = fvalue;
    }
    count[offset] = lcount;
}

__global__ static void materialization_index(int* dim, int * dim_attr, int* psum, int* fact, int* fact_attr, long inNum, int * factFilter, int * result1, int * result2, int * result3)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;
    int localCount = psum[offset];

    for(int i=offset;i<inNum;i+=stride){
    	int dimID = factFilter[i];
    	if(dimID != 0)
    	{    		
    		((int*)result1)[localCount] = ((int *)fact)[i];
    		((int*)result2)[localCount] = ((int *)fact_attr)[i];
    		((int*)result3)[localCount] = ((int *)dim_attr)[dimID];
            localCount ++;
    	}
    }
}

__global__ static void materialization(int* dim, int * dim_attr, int* psum, int* fact, int* fact_attr, long inNum, int * factFilter, int * result1, int * result2, int * result3)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;
    int localCount = psum[offset];

    for(int i=offset;i<inNum;i+=stride){
    	int dimID = factFilter[i];
    	if(dimID != 0)
    	{    		
    		((int*)result1)[localCount] = ((int *)fact)[i];
    		((int*)result2)[localCount] = ((int *)fact_attr)[i];
    		((int*)result3)[localCount] = ((int *)dim_attr)[dimID - 1];
            localCount ++;
    	}
    }
}

__global__ static void scanCol(int inNum, int * result1, int * result2, int * result3, bool *mapbit)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){
    	if(result2[i] == result3[i])
    	{
    		mapbit[i] = true;
    	}
    }
}


__global__ static void build_hash_table_dual(int *dim, int * dim_index, long inNum, int *psum, int * bucket,int hsize){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){
        int joinKey = ((int *) dim)[i]; 
        int joinAttr = ((int *) dim_index)[i]; 
        int hKey = joinKey & (hsize-1);
        int pos = atomicAdd(&psum[hKey],1) * 3;
        assert(pos < inNum * 3);
        ((int*)bucket)[pos] = joinKey;
        pos += 1;
        ((int*)bucket)[pos] = joinAttr;
        pos += 1;
        int dimId = i+1;
        ((int*)bucket)[pos] = dimId;
    }
}

__global__ static void count_join_result_dual(int* num, int* psum, int* bucket, int* fact, int * fact_attr, long inNum, bool* count,int hsize){

    int lcount = 0;
    int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){

        int fkey = ((int *)(fact))[i];
        int hkey = fkey &(hsize-1);
        int keyNum = num[hkey];
        int fvalue = 0;

        for(int j=0;j<keyNum;j++){
            int pSum = psum[hkey];
            int dimKey = ((int *)(bucket))[3*j + 3*pSum];

            if(dimKey == fkey){
                int dimAttr = ((int *)(bucket))[3*j + 3*pSum + 1];
                int factAttr = ((int *)(fact_attr))[i];
                if(dimAttr == factAttr)
                {
                	count[i] = true;
                	break;
            	}
            }
        }
    }    
}


__global__ static void build_hash_table_nest(int *dim, int * dim_index, long inNum, int *psum, int * bucket,int hsize){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){
        int joinKey = ((int *) dim)[i]; 
        int joinAttr = ((int *) dim_index)[i]; 
        int hKey = joinKey & (hsize-1);
        int pos = atomicAdd(&psum[hKey],1) * 2;
        assert(pos < inNum * 3);
        ((int*)bucket)[pos] = joinKey;
        pos += 1;
        ((int*)bucket)[pos] = joinAttr;
    }
}

__global__ static void count_join_result_nest(int* num, int* psum, int* bucket, int* fact, int * fact_attr, long inNum, bool* count,int hsize){

    int lcount = 0;
    int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){

        int fkey = ((int *)(fact))[i];
        int hkey = fkey &(hsize-1);
        int keyNum = num[hkey];
        int fvalue = 0;

        for(int j=0;j<keyNum;j++){
            int pSum = psum[hkey];
            int dimKey = ((int *)(bucket))[2*j + 2*pSum];

            if(dimKey == fkey){
                int dimAttr = ((int *)(bucket))[2*j + 2*pSum + 1];
                int factAttr = ((int *)(fact_attr))[i];
                if(dimAttr == factAttr)
                {
                	count[i] = true;
                	break;
            	}
            }
        }
    }    
}

__global__ static void compare(int * a, int *b, int size)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<size;i+=stride){
    	if(a[i] != b[i])
    	{
    		printf("error");
    		assert(a[i] == b[i]);
    	}
    }
}
template<typename vec_t>
__global__ static void reorganize(vec_t * key, vec_t * index, int inNum, vec_t * bucket)
{
	int stride = blockDim.x * gridDim.x;
    long offset = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=offset;i<inNum;i+=stride){
    	bucket[i*2] = key[i];
    	bucket[i*2+1] = index[i];
    }
}



__global__ static void nestScanFilterDistributedCorrelated_deviceFunc(//Sofokils function
     int *col, int * innerTableSecondLinkingPredicate_d,  long tupleNum, // Inner table
     int * outerTableData, int * outerTableSecondLinkingPredicate_d, int outerTableDataRows, //Outer Table
     int * num, int * psum, int hsize,
     int innerThreadGpuMapping, //Inner mapping
     bool * innerOuterMatchingBitmap) //Output
    {

    //Need to think how we are going to compare the second predicate

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con, outerValue, outerValue2, innerValue, innerValue2;
    int indexKey, indexNum, indexPos ;


    if(innerThreadGpuMapping == 0) // Sofoklis default mapping
    {
	    for (long j=0; j<outerTableDataRows;j++){
	        
		    /* Get outer table values */
		    outerValue = outerTableData[j];
		    outerValue2 = outerTableSecondLinkingPredicate_d[j];
		    
		    /* Filter result */
		    for(long i = tid; i<tupleNum;i+=stride){
		    
		        //Get inner values
		        innerValue = ((int*)col)[i];
		        innerValue2 = ((int*)innerTableSecondLinkingPredicate_d)[i];

		        /* Store bool value (only threads that have a match) */
		        if (outerValue == innerValue && outerValue2 == innerValue2){
		            innerOuterMatchingBitmap[j]=true;
		            break;
		        }
		    }
		}
	}
	else //sort index scan
	{
		for (long j=tid; j<outerTableDataRows;j+=stride){	        
		    /* Get outer table values */
		    outerValue = outerTableData[j];
		    outerValue2 = outerTableSecondLinkingPredicate_d[j];

		    indexKey   = outerValue & (hsize -1);
		    indexNum   = num[indexKey];
		    indexPos   = psum[indexKey];

		    /* Filter result */
		    for(long i = 0; i<indexNum;i++){
		    
		        //Get inner values
		        innerValue = ((int*)col)[i + indexPos];
		        innerValue2 = ((int*)innerTableSecondLinkingPredicate_d)[i + indexPos];

		        /* Store bool value (only threads that have a match) */
		        if (outerValue == innerValue && outerValue2 == innerValue2){
		            innerOuterMatchingBitmap[j]=true;
		            break;
		        }
		    }
		}
	}
}


__global__ static void unnestScanFilterDistributedCorrelated_deviceFunc(
     int *col, int * innerTableSecondLinkingPredicate_d,  long tupleNum, // Inner table
     int * index, // index for inner table
     int * outerTableData, int * outerTableSecondLinkingPredicate_d, int outerTableDataRows, //Outer Table
     int * num, int * psum, int hsize,
     int innerThreadGpuMapping, //Inner mapping
     bool * innerOuterMatchingBitmap) //Output
    {

    //Need to think how we are going to compare the second predicate

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con, outerValue, outerValue2, innerValue, innerValue2;
    int indexKey, indexNum, indexPos ;

    if(innerThreadGpuMapping == 0)
    {
	    for (long j=0; j<outerTableDataRows;j++){
	        
		    /* Get outer table values */
		    outerValue = outerTableData[j];
		    outerValue2 = outerTableSecondLinkingPredicate_d[j];
		    
		    /* Filter result */
		    for(long i = tid; i<tupleNum;i+=stride){
		    
		        //Get inner values
		        innerValue = ((int*)col)[i];
		        innerValue2 = ((int*)innerTableSecondLinkingPredicate_d)[i];

		        /* Store bool value (only threads that have a match) */
		        if (outerValue == innerValue && outerValue2 == innerValue2){
		            innerOuterMatchingBitmap[j]=true;
		            break;
		        }
		    }
		}
	}
	else
	{
		for (long j=tid; j<outerTableDataRows;j+=stride){	        
		    /* Get outer table values */
		    outerValue = outerTableData[j];
		    outerValue2 = outerTableSecondLinkingPredicate_d[j];

		    indexKey   = outerValue & (hsize -1);
		    indexNum   = num[indexKey];
		    indexPos   = psum[indexKey];

		    /* Filter result */
		    for(long i = 0; i<indexNum;i++){
		    	int target = index[i + indexPos];
		        //Get inner values
		        innerValue = ((int*)col)[target];
		        innerValue2 = ((int*)innerTableSecondLinkingPredicate_d)[target];

		        /* Store bool value (only threads that have a match) */
		        if (outerValue == innerValue && outerValue2 == innerValue2){
		            innerOuterMatchingBitmap[j]=true;
		            break;
		        }
		    }
		}
	}
}

__global__ static void recallScan(
     int * col, int * innerTableSecondLinkingPredicate_d, int tupleNum,// Inner table
     int outerValue, int outerValue2, int offset, //Outer Table
     bool * innerOuterMatchingBitmap) //Output
{

	int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con, innerValue, innerValue2;

	for(long i = tid; i<tupleNum;i+=stride){
		    
        //Get inner values
        innerValue = ((int*)col)[i];
        innerValue2 = ((int*)innerTableSecondLinkingPredicate_d)[i];

        /* Store bool value (only threads that have a match) */
        if (outerValue == innerValue && outerValue2 == innerValue2){
            innerOuterMatchingBitmap[offset]=true;
            break;
        }
    }
} 

__global__ static void nestScanDynamic_deviceFunc(
     int *col, int * innerTableSecondLinkingPredicate_d,  long tupleNum, // Inner table
     int * outerTableData, int * outerTableSecondLinkingPredicate_d, int outerTableDataRows, //Outer Table
     int innerThreadGpuMapping, //Inner mapping
     bool * innerOuterMatchingBitmap) //Output
    {

    //Need to think how we are going to compare the second predicate

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con, outerValue, outerValue2, innerValue, innerValue2;
    int indexKey, indexNum, indexPos ;

    if(innerThreadGpuMapping == 0)
    {
	    for (long j=tid; j<outerTableDataRows;j+=stride){
	        
		    /* Get outer table values */
		    outerValue = outerTableData[j];
		    outerValue2 = outerTableSecondLinkingPredicate_d[j];

		    recallScan<<<4096,256>>>((int*)col,(int*)innerTableSecondLinkingPredicate_d,tupleNum,outerValue,outerValue2,j,innerOuterMatchingBitmap);
		    
		    __syncthreads();
		    // /* Filter result */
		    // for(long i = tid; i<tupleNum;i+=stride){
		    
		    //     //Get inner values
		    //     innerValue = ((int*)col)[i];
		    //     innerValue2 = ((int*)innerTableSecondLinkingPredicate_d)[i];

		    //     /* Store bool value (only threads that have a match) */
		    //     if (outerValue == innerValue && outerValue2 == innerValue2){
		    //         innerOuterMatchingBitmap[j]=true;
		    //         break;
		    //     }
		    // }
		}
		// hipDeviceSynchronize();
	}
}

__global__ static void recallScanUmlimited(
     int outerValue, int * allCol, int innerIndex, int outerIndex, int currentLevel, 
     int loopLevel, int * attrSize, int * attrNum, int * round, bool * innerOuterMatchingBitmap) //Output
{

	int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con, innerValue;

    //Get inner values
    int index = innerIndex - attrNum[0];
    int tableIndex, tableBegin;
    if(index >= 2)
    {
    	index -= 2;
    	tableIndex = 2;
    	tableBegin = attrNum[0] * attrSize[0] + attrNum[1] * attrSize[1];
    }
    else
    {
    	tableIndex = 1;
    	tableBegin = attrNum[0] * attrSize[0];
    }
    int tupleNum = attrSize[tableIndex];

	for(long i = tid; i<tupleNum;i+=stride){		    
        
        innerValue = allCol[tableBegin + tupleNum * index + i];

        /* Store bool value (only threads that have a match) */
        if (outerValue == innerValue){
        	if(currentLevel == loopLevel)
        	{
            	innerOuterMatchingBitmap[outerIndex] = true;
            	return;
        	}
            else
            {
            	int inner1 = round[currentLevel * 2 + 1];
            	int outer1 = round[currentLevel * 2];

            	int outerValue1 = allCol[outer1 * attrSize[0] + outerIndex];
            	recallScanUmlimited<<<4096,256>>>(outerValue1, allCol, inner1, outerIndex, currentLevel + 1, loopLevel, attrSize, attrNum, round,innerOuterMatchingBitmap);
            }
        }
    }
} 


__global__ static void nestScanDynamicUnlimited_deviceFunc(
     int * allCol, int * nuknow, int loopLevel, int * attrSize, int * attrNum, int * round,
     int innerThreadGpuMapping, //Inner mapping
     bool * innerOuterMatchingBitmap) //Output
    {

    //Need to think how we are going to compare the second predicate

    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int con, outerValue, outerValue2, innerValue, innerValue2;
    int indexKey, indexNum, indexPos ;

    if(innerThreadGpuMapping == 0)
    {
	    for (long j=tid; j<attrSize[0];j+=stride){
	        
	        int r0 = round[0];
		    /* Get outer table values */
		    outerValue = allCol[r0 * attrSize[0] + j];
		    int r1 = round[1];
		    recallScanUmlimited<<<2048,128>>>(outerValue, allCol, r1, j, 1, loopLevel, attrSize, attrNum, round,innerOuterMatchingBitmap);
		    
		    __syncthreads();

		}
	}
}

hipError_t Subquery_proc(int *dim,int *dim_attr,int *dim_index,int lsize,int *fact,int *fact_attr,int* fact_index,int rsize)
{
    struct timespec start_t, end_t;
    int defaultBlock = 4096;
    double total = 0.0;
    double timeE = 0.0;
    hipStream_t s; 
    CUDACHECK(hipStreamCreate(&s)); 

    dim3 grid(defaultBlock);
    dim3 block(256);
    int myRank = 0;
    int hsize = lsize;
    NP2(hsize);

    int *gpu_dim_psum = NULL, *gpu_fact_psum = NULL, *gpu_dim_hashNum = NULL;
    int total_count = 0;
    int all_count = 0;
    int * d_max, max, mask,*gpu_count_db = NULL,*gpu_count_new = NULL;
    int alloc_size = rsize > lsize ? rsize : lsize;
    int * factFilter_db = NULL, * bucket = NULL; 
    int * filterResult1 = NULL, * filterResult2 = NULL, * filterResult3 = NULL, * gpu_resPsum = NULL;
    bool * mapbit =NULL;
    clock_gettime(CLOCK_REALTIME,&start_t);
    
    CUDACHECK(hipMalloc((void **)&gpu_dim_hashNum, hsize * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&gpu_dim_psum, hsize * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&gpu_fact_psum, hsize * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&d_max, sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&gpu_count_db, 4096*256*sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&gpu_resPsum, 4096*256*sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&factFilter_db, rsize * sizeof(int))); 
    CUDACHECK(hipMemset(gpu_dim_psum,0, hsize * sizeof(int)));
    CUDACHECK(hipMemset(gpu_fact_psum,0, hsize * sizeof(int)));

    CUDACHECK(hipMalloc((void **)&mapbit, lsize * sizeof(bool)));    

    CUDACHECK(hipMalloc((void **)&bucket, lsize * 3 * sizeof(int))); 

    hipMemset(mapbit, 0, lsize * sizeof(bool));

    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
    printf("init Time: %lf ms hsize %d full block size %d\n", timeE/(1000*1000),hsize,grid.x);
    

    clock_gettime(CLOCK_REALTIME,&start_t);
    CUDACHECK(hipMemset(gpu_dim_hashNum,0, hsize * sizeof(int)));  
    CUDACHECK(hipMemset(factFilter_db,0, rsize * sizeof(int)));    
    CUDACHECK(hipMemset(gpu_count_db,0, 4096*256 * sizeof(int)));   
    
    count_hash_num<<<4096,256>>>(dim,lsize,gpu_dim_hashNum,hsize);
    CUDACHECK(hipDeviceSynchronize());
    thrust::exclusive_scan(thrust::device, gpu_dim_hashNum, gpu_dim_hashNum + hsize, gpu_dim_psum); // in-place scan

    hipMemcpy(gpu_fact_psum, gpu_dim_psum, hsize * sizeof(int), hipMemcpyDeviceToDevice);

    build_hash_table<<<4096,256>>>(dim,lsize,gpu_fact_psum,bucket,hsize);

    CUDACHECK(hipDeviceSynchronize());
    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
    printf("build Time: %lf ms \n",  timeE/(1000*1000));
    total += timeE;   

    clock_gettime(CLOCK_REALTIME,&start_t);

    count_join_result<<<4096,256>>>(gpu_dim_hashNum,gpu_dim_psum,bucket,fact,rsize,gpu_count_db,factFilter_db,hsize);
    int tmp1, tmp2;
    CUDACHECK(hipDeviceSynchronize());
    see<<<1,1>>>(gpu_count_db);
    CUDACHECK(hipMemcpy(&tmp1,&gpu_count_db[4096*256-1],sizeof(int),hipMemcpyDeviceToHost));
    thrust::exclusive_scan(thrust::device, gpu_count_db, gpu_count_db + 4096*256, gpu_resPsum); 
    CUDACHECK(hipMemcpy(&tmp2,&gpu_resPsum[4096*256-1],sizeof(int),hipMemcpyDeviceToHost));

    int resCount = tmp1 + tmp2;

    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
    printf("%d rows probe Time: %lf ms \n", resCount, timeE/(1000*1000));
    total += timeE;   

    clock_gettime(CLOCK_REALTIME,&start_t);

    CUDACHECK(hipMalloc((void **)&filterResult1, resCount * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&filterResult2, resCount * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&filterResult3, resCount * sizeof(int))); 

    materialization<<<4096,256>>>(dim, dim_attr, gpu_resPsum, fact, fact_attr, rsize, factFilter_db, filterResult1,filterResult2,filterResult3);
    CUDACHECK(hipDeviceSynchronize());
    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
    printf("materialization Time: %lf ms \n",  timeE/(1000*1000));
    total += timeE;  

    clock_gettime(CLOCK_REALTIME,&start_t);

    scanCol<<<4096,256>>>(resCount, filterResult1,filterResult2,filterResult3,mapbit);

    CUDACHECK(hipDeviceSynchronize());
    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
    printf("scan Time: %lf ms \n", timeE/(1000*1000));
    total += timeE;   

    hipFree(filterResult1);
    hipFree(filterResult2);
    hipFree(filterResult3);

    printf("GPU-DB unnest Time: %lf ms\n\n", total/(1000*1000));

    int * allAttr = NULL, * roundAttr = NULL, * d_attrSize = NULL, * d_attrNum = NULL, * d_roundNum;
    //three table size
    int attrSize[3] = {lsize, rsize, rsize};
    //number of attrs in each table
    int attrNum[3] = {3, 2, 2};    
    int roundNum[6] = {0, 3 , 1, 4, 0, 3};
    // indicate the comaprison , even number only, three loops
    //e.g., the first number of 2 indicate the first two attrs in roundNum will be compared
    int roundHist[3] = {2,2,2}; 
    int roundPsum[3] = {0,2,4};
    // only two loops now
    int roundLevel = 2;

    int attrTotalSize = 0;
    attrTotalSize = attrNum[0] * attrSize[0] + attrNum[1] * attrSize[1] + attrNum[2] * attrSize[2];

    CUDACHECK(hipMalloc((void **)&allAttr, attrTotalSize * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&roundAttr, 3 * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&d_attrSize, 3 * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&d_attrNum, 3 * sizeof(int))); 
    CUDACHECK(hipMalloc((void **)&d_roundNum, 6 * sizeof(int))); 

    CUDACHECK(hipMemcpy(d_attrSize,attrSize, 3 * sizeof(int),hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(d_attrNum,attrNum, 3 * sizeof(int),hipMemcpyHostToDevice));

    hipMemcpy(d_roundNum,roundNum, 6 * sizeof(int),hipMemcpyHostToDevice); 

    attrTotalSize = 0;

    hipMemcpy(allAttr,dim, lsize * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(allAttr + lsize,dim_attr, lsize * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(allAttr + 2 * lsize,dim_index, lsize * sizeof(int), hipMemcpyDeviceToDevice);

    attrTotalSize += attrNum[0] * attrSize[0];

    hipMemcpy(allAttr + attrTotalSize,fact, rsize * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(allAttr + attrTotalSize + rsize,fact_attr, rsize * sizeof(int), hipMemcpyDeviceToDevice);

    attrTotalSize += attrNum[1] * attrSize[1];

    hipMemcpy(allAttr + attrTotalSize,fact_index, rsize * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(allAttr + attrTotalSize + rsize,fact, rsize * sizeof(int), hipMemcpyDeviceToDevice);    

    clock_gettime(CLOCK_REALTIME,&start_t);

    hipMemset(mapbit, 0, lsize * sizeof(bool));

    nestScanDynamicUnlimited_deviceFunc<<<32,128>>>(allAttr,roundAttr,roundLevel,d_attrSize,d_attrNum,d_roundNum,0,mapbit);

    CUDACHECK(hipDeviceSynchronize());
    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
    printf("dynamic scan Time: %lf ms \n", timeE/(1000*1000));
    total += timeE; 

    hipError_t cudaStatus = hipDeviceSynchronize();    

    if(cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Subquery error\n");
        CUDACHECK(cudaStatus);
    }  


    total += timeE;

    clock_gettime(CLOCK_REALTIME, &start_t);
    // CUDACHECK(hipFree(gpu_data));
    // CUDACHECK(hipFree(gpu_index));
    CUDACHECK(hipFree(gpu_dim_hashNum));
    CUDACHECK(hipFree(gpu_dim_psum));
    CUDACHECK(hipFree(gpu_fact_psum));
    clock_gettime(CLOCK_REALTIME,&end_t);
    timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;

    printf("local mem free Time: %lf ms\n", timeE/(1000*1000));
    total += timeE;

    cudaStatus = hipDeviceSynchronize();
    CUDACHECK(cudaStatus);
    return cudaStatus;
}


hipError_t Preparation(int *all_l_table, int *all_l_attr, int l_size, int *all_r_table, int *all_r_attr, int r_size, int argc, char* argv[]);

int main(int argc, char* argv[])
{
	int a_size = 4*1024*1024;
	int b_size = 1*1024*1024;

	if(argc >= 2)
	{
		if(atoi(argv[1]) != 0)
		{
			a_size = atoi(argv[1])*1024*1024;
		}
	}

	if(argc >= 3)
	{
		if(atoi(argv[2])!=0)
			b_size = atoi(argv[2])*1024*1024;
	}
	int *a,*a_1;
	int *b,*b_1;

	fprintf(stderr,"R relation size %d Rows , S relation size %d Rows bit %d\n",a_size,b_size);

	CUDACHECK(hipHostAlloc((void **)&a, sizeof(int)*a_size, hipHostMallocPortable | hipHostMallocMapped));
	CUDACHECK(hipHostAlloc((void **)&b, sizeof(int)*b_size, hipHostMallocPortable | hipHostMallocMapped));
	CUDACHECK(hipHostAlloc((void **)&a_1, sizeof(int)*a_size, hipHostMallocPortable | hipHostMallocMapped));
	CUDACHECK(hipHostAlloc((void **)&b_1, sizeof(int)*b_size, hipHostMallocPortable | hipHostMallocMapped));

	for (int i = 0; i < b_size; i++)
	{
		int tmp = rand()%a_size;
		if(!tmp)
			tmp ++;
		b[i] = tmp;	
		b_1[i] = b_size - i;	
	}
	for (int i = 0; i < a_size; i++)
	{
		int tmp = rand()%a_size;
		if(!tmp)
			tmp ++;
		a[i] = tmp;
		a_1[i] = a_size - i;	
	}
	hipError_t cudaStatus;

	cudaStatus = Preparation(a, a_1, a_size,b, b_1, b_size, argc, argv);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Subquery failed!\n");
		return 1;
	}

	return 0;
}

template<typename vec_t>
__global__ static void assign_index(vec_t *dim, long  inNum,int rank){
    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = offset; i<inNum; i += stride)
        dim[i] = i;
}

hipError_t Preparation(int *all_l_table, int *all_l_index, int l_size, int *all_r_table, int *all_r_index, int r_size, int argc, char* argv[])
{
	struct timespec start, end;
	struct timespec start_t, end_t;
	hipError_t cudaStatus;
	hipEvent_t event;
	hipIpcMemHandle_t memHandle_t;

	int defaultBlock = 4096;	
	int new_size = 0;
	dim3 grid(defaultBlock);
	dim3 block(256);
	hipStream_t s; 
	CUDACHECK(hipStreamCreate(&s)); 

	double total = 0.0;

	clock_gettime(CLOCK_REALTIME,&start);
	int * gpu_dim     = NULL, * gpu_dim_index = NULL, * gpu_fact  = NULL, *gpu_fact_index = NULL;
	int * gpu_dim_attr= NULL, * gpu_fact_attr = NULL;

	clock_gettime(CLOCK_REALTIME,&start_t);
	int primaryKeySize = r_size * sizeof(int);
	int filterSize = l_size * sizeof(int);

	CUDACHECK(hipMalloc((void **)&gpu_fact, primaryKeySize));
	CUDACHECK(hipMalloc((void **)&gpu_fact_attr, primaryKeySize));
	CUDACHECK(hipMalloc((void **)&gpu_dim, filterSize));
	CUDACHECK(hipMalloc((void **)&gpu_dim_attr, filterSize));
	CUDACHECK(hipMalloc((void **)&gpu_fact_index, primaryKeySize));
	CUDACHECK(hipMalloc((void **)&gpu_dim_index, filterSize));

	clock_gettime(CLOCK_REALTIME,&end_t);
	double timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
	total += timeE;

	printf("init Time: %lf ms\n", timeE/(1000*1000));	

	clock_gettime(CLOCK_REALTIME,&start_t);
	CUDACHECK(hipMemcpyAsync(gpu_dim, all_l_table, sizeof(int)*l_size, hipMemcpyHostToDevice, s));
	CUDACHECK(hipMemcpyAsync(gpu_fact, all_r_table , sizeof(int)*r_size, hipMemcpyHostToDevice, s));
	CUDACHECK(hipMemcpyAsync(gpu_dim_attr, all_l_index, sizeof(int)*l_size, hipMemcpyHostToDevice, s));
	CUDACHECK(hipMemcpyAsync(gpu_dim_attr, all_r_index , sizeof(int)*r_size, hipMemcpyHostToDevice, s));
	assign_index<int><<<grid,block>>>(gpu_dim_index,l_size,0);
	assign_index<int><<<grid,block>>>(gpu_fact_index,r_size,0);
	CUDACHECK(hipDeviceSynchronize());

	clock_gettime(CLOCK_REALTIME,&end_t);
	timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
	total += timeE;
	int recv_offset = 0;

	printf("Host To Device Time: %lf ms\n", timeE/(1000*1000));

	clock_gettime(CLOCK_REALTIME, &start_t);

	cudaStatus = Subquery_proc(gpu_dim,gpu_dim_attr,gpu_dim_index,l_size,gpu_fact,gpu_dim_attr,gpu_fact_index,r_size);
	
	clock_gettime(CLOCK_REALTIME,&end_t);
	timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
	total += timeE;

	printf("Subquery Time: %lf ms\n", timeE/(1000*1000));
	
	clock_gettime(CLOCK_REALTIME, &start_t);
	CUDACHECK(hipFree(gpu_fact));
	CUDACHECK(hipFree(gpu_fact_index));
	CUDACHECK(hipFree(gpu_dim));
	CUDACHECK(hipFree(gpu_dim_index));

	clock_gettime(CLOCK_REALTIME,&end_t);
	timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
	printf("second GPU original memory free Time: %lf ms\n", timeE/(1000*1000));
	total += timeE;

	clock_gettime(CLOCK_REALTIME, &start_t);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME,&end_t);
	timeE = (end_t.tv_sec -  start_t.tv_sec)* BILLION + end_t.tv_nsec - start_t.tv_nsec;
	printf("second CPU memory free Time: %lf ms\n", timeE/(1000*1000));
	total += timeE;

	clock_gettime(CLOCK_REALTIME,&end);
	timeE = (end.tv_sec -  start.tv_sec)* BILLION + end.tv_nsec - start.tv_nsec;
	printf("Whole Processing Time: %lf ms Whole time : %1f ms \n", total/(1000*1000),timeE/(1000*1000));
		
	return hipDeviceSynchronize();
}