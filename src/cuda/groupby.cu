#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <time.h>
#include "../include/common.h"
#include "../include/gpuCudaLib.h"
#include "../include/cudaHash.h"
#include "../include/mempool.h"
#include "scanImpl.cu"

/*
 * Combine the group by columns to build the group by keys. 
 */

__global__ static void build_groupby_key(char ** content, int gbColNum, int * gbIndex, int * gbType, int * gbSize, long tupleNum, int * key, int *num, int* groupNum){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(long i = offset; i< tupleNum; i+= stride){
        char buf[128] = {0};
        for (int j=0; j< gbColNum; j++){
            char tbuf[32]={0};
            int index = gbIndex[j];

            if (index == -1){
                gpuItoa(1,tbuf,10);
                gpuStrncat(buf,tbuf,1);

            }else if (gbType[j] == STRING){
                gpuStrncat(buf, content[index] + i*gbSize[j], gbSize[j]);

            }else if (gbType[j] == INT){
                int key = ((int *)(content[index]))[i];
                gpuItoa(key,tbuf,10);
                gpuStrcat(buf,tbuf);
            }
        }
        int hkey = StringHash(buf) % HSIZE;
        key[i]= hkey;
        num[hkey] = 1;
        atomicAdd(&(groupNum[hkey]), 1);
    }
}


/*
 * Count the number of groups 
 */

__global__ static void count_group_num(int *num, int tupleNum, int *totalCount){
        int stride = blockDim.x * gridDim.x;
        int offset = blockIdx.x * blockDim.x + threadIdx.x;
        int localCount = 0;

        for(int i=offset; i<tupleNum; i+= stride){
                if(num[i] == 1){
                        localCount ++;
                }
        }

        atomicAdd(totalCount,localCount);
}

/*
 * Calculate the groupBy expression.
 */

__device__ static float calMathExp(char **content, struct mathExp exp, int pos){
    float res ;

    if(exp.op == NOOP){
        if (exp.opType == CONS)
            res = exp.opValue;
        else if(exp.opType == COLUMN_INTEGER){
            int index = exp.opValue;
            res = ((int *)(content[index]))[pos];
        }else if(exp.opType == COLUMN_DECIMAL){
            int index = exp.opValue;
            res = ((float *)(content[index]))[pos];
        }
    
    }else if(exp.op == PLUS ){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) + calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == MINUS){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) - calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == MULTIPLY){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) * calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == DIVIDE){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) / calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);
    }

    return res;
}


__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
        float old;
        old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
             __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

        return old;
}

/*
 * group by constant. Currently only support SUM function.
 */

__global__ static void agg_cal_cons(char ** content, int colNum, struct groupByExp* exp, long tupleNum, char ** result){

    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float buf[32];
    for(int i = 0; i < colNum; i++){
        int func = exp[i].func;
        if(func == MAX)
            buf[i] = FLOAT_MIN;
        else if(func == MIN)
            buf[i] = FLOAT_MAX;
        else
            buf[i] = 0;
    }

    for(int i=index;i<tupleNum;i+=stride){
        for(int j=0;j<colNum;j++){
            int func = exp[j].func;
            if (func == SUM){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                buf[j] += tmpRes;
            }else if (func == AVG){

                float tmpRes = calMathExp(content, exp[j].exp, i)/tupleNum;
                buf[j] += tmpRes;
            }else if (func == MAX){

                float tmpRes = calMathExp(content, exp[j].exp, i);
                buf[j] = buf[j] > tmpRes ? buf[j] : tmpRes;
            }else if (func == MIN){

                float tmpRes = calMathExp(content, exp[j].exp, i);
                buf[j] = buf[j] < tmpRes ? buf[j] : tmpRes;
            }
        }
    }

    for(int i=0;i<colNum;i++)
    {
        int func = exp[i].func;
        if (func == SUM)
            atomicAdd(&((float *)result[i])[0], buf[i]);
        else if (func == MAX)
            atomicMaxFloat(&((float *)result[i])[0], buf[i]);
        else if (func == MIN)
            atomicMinFloat(&((float *)result[i])[0], buf[i]);
    }
}

/*
 * gropu by
 */

__global__ static void agg_cal(char ** content, int colNum, struct groupByExp* exp, int * gbType, int * gbSize, long tupleNum, int * key, int *psum, int * groupNum, char ** result){

    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=index;i<tupleNum;i+=stride){

        int hKey = key[i];
        int offset = psum[hKey];

        for(int j=0;j<colNum;j++){
            int func = exp[j].func;
            if(func ==NOOP){
                int type = exp[j].exp.opType;

                if(type == CONS){
                    int value = exp[j].exp.opValue;
                    ((int *)result[j])[offset] = value;
                }else{
                    int index = exp[j].exp.opValue;
                    int attrSize = gbSize[j];
                    if(attrSize == sizeof(int))
                        ((int *)result[j])[offset] = ((int*)content[index])[i];
                    else
                        memcpy(result[j] + offset*attrSize, content[index] + i * attrSize, attrSize);
                }

            }else if (func == SUM ){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                atomicAdd(& ((float *)result[j])[offset], tmpRes);
            }else if (func == MAX ){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                atomicMaxFloat(& ((float *)result[j])[offset], tmpRes);

            }else if (func == MIN ){
                float tmpRes = calMathExp(content, exp[j].exp, i);
                atomicMinFloat(& ((float *)result[j])[offset], tmpRes);

            }else if (func == AVG){
                float tmpRes = calMathExp(content, exp[j].exp, i)/groupNum[hKey];
                atomicAdd(& ((float *)result[j])[offset], tmpRes);
            }
        }
    }
}

__global__ static void init_int_array(int *array, int array_size, int init_value)
{
    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = index; i < array_size; i += stride)
        array[i] = init_value;
}
/* 
 * groupBy: group by the data and calculate. 
 * 
 * Prerequisite:
 *  input data are not compressed
 *
 * Input:
 *  gb: the groupby node which contains the input data and groupby information
 *  pp: records the statistics such as kernel execution time 
 *
 * Return:
 *  a new table node
 */

struct tableNode * groupBy(struct groupByNode * gb, struct statistic * pp, const bool use_mempool = false, const bool use_gpu_mempool = false){

    struct timespec start,end;
    clock_gettime(CLOCK_REALTIME,&start);
    int *gpuGbIndex = NULL, gpuTupleNum, gpuGbColNum;
    int *gpuGbType = NULL, *gpuGbSize = NULL;

    int *gpuGbKey = NULL;
    char ** gpuContent = NULL, **column = NULL;

    /*
     * @gbCount: the number of groups
     * gbConstant: whether group by constant
     */

    int gbCount;
    int gbConstant = 0;

    struct tableNode *res = NULL;
    if(!use_mempool) {
        res = (struct tableNode *) malloc(sizeof(struct tableNode));
        CHECK_POINTER(res);
    } else {
        res = (struct tableNode *) alloc_mempool(sizeof(struct tableNode));
        MEMPOOL_CHECK();
    }

    res->tupleSize = gb->tupleSize;
    res->totalAttr = gb->outputAttrNum;

    if(!use_mempool) {
        res->attrType = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrType);
        res->attrSize = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrSize);
        res->attrTotalSize = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->attrTotalSize);
        res->dataPos = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->dataPos);
        res->dataFormat = (int *) malloc(sizeof(int) * res->totalAttr);
        CHECK_POINTER(res->dataFormat);
        res->content = (char **) malloc(sizeof(char **) * res->totalAttr);
        CHECK_POINTER(res->content);
    } else {
        res->attrType = (int *) alloc_mempool(sizeof(int) * res->totalAttr);
        res->attrSize = (int *) alloc_mempool(sizeof(int) * res->totalAttr);
        res->attrTotalSize = (int *) alloc_mempool(sizeof(int) * res->totalAttr);
        res->dataPos = (int *) alloc_mempool(sizeof(int) * res->totalAttr);
        res->dataFormat = (int *) alloc_mempool(sizeof(int) * res->totalAttr);
        res->content = (char **) alloc_mempool(sizeof(char **) * res->totalAttr);
        MEMPOOL_CHECK();
    }

    for(int i=0;i<res->totalAttr;i++){
        res->attrType[i] = gb->attrType[i];
        res->attrSize[i] = gb->attrSize[i];
        res->dataFormat[i] = UNCOMPRESSED;
    }
    
    gpuTupleNum = gb->table->tupleNum;
    gpuGbColNum = gb->groupByColNum;

    if(gpuGbColNum == 1 && gb->groupByIndex[0] == -1){
        gbConstant = 1;
    }


    dim3 grid(1024);
    dim3 block(128);
    int blockNum = gb->table->tupleNum / block.x + 1;
    if(blockNum < 1024)
        grid = blockNum;

    int *gpu_hashNum = NULL, *gpu_psum = NULL, *gpuGbCount = NULL, *gpu_groupNum = NULL;

    if(!use_gpu_mempool) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuContent, gb->table->totalAttr * sizeof(char *)));
    } else {
        alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuContent, gb->table->totalAttr * sizeof(char *));
        GPU_MEMPOOL_CHECK(gpu_inner_mp);
    }

    if(!use_mempool) {
        column = (char **) malloc(sizeof(char *) * gb->table->totalAttr);
        CHECK_POINTER(column);
    } else {
        column = (char **) alloc_mempool(sizeof(char *) * gb->table->totalAttr);
        MEMPOOL_CHECK();
    }

    for(int i=0;i<gb->table->totalAttr;i++){
        int attrSize = gb->table->attrSize[i];
        if(gb->table->dataPos[i]==MEM){
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)& column[i], attrSize * gb->table->tupleNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[i], gb->table->content[i], attrSize *gb->table->tupleNum, hipMemcpyHostToDevice));

            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &column[i], sizeof(char *), hipMemcpyHostToDevice));
        }else{
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &gb->table->content[i], sizeof(char *), hipMemcpyHostToDevice));
        }
    }

    if(gbConstant != 1){

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int) * gb->groupByColNum));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbType, sizeof(int) * gb->groupByColNum);
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType,gb->groupByType, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));
        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int) * gb->groupByColNum));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbSize, sizeof(int) * gb->groupByColNum);
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbSize,gb->groupBySize, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));


        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbKey, gb->table->tupleNum * sizeof(int)));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbKey, sizeof(int) * gb->groupByColNum);
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbIndex, sizeof(int) * gb->groupByColNum));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbIndex, sizeof(int) * gb->groupByColNum);
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbIndex, gb->groupByIndex,sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_hashNum,sizeof(int)*HSIZE));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpu_hashNum, sizeof(int) * HSIZE);
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_hashNum,0,sizeof(int)*HSIZE));

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_groupNum,sizeof(int)*HSIZE));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpu_groupNum, sizeof(int) * HSIZE);
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_groupNum,0,sizeof(int)*HSIZE));

        build_groupby_key<<<grid,block>>>(gpuContent,gpuGbColNum, gpuGbIndex, gpuGbType,gpuGbSize,gpuTupleNum, gpuGbKey, gpu_hashNum, gpu_groupNum);
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbIndex));
        }

        gbCount = 1;

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbCount,sizeof(int)));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbCount, sizeof(int));
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuGbCount, 0, sizeof(int)));

        count_group_num<<<grid,block>>>(gpu_hashNum, HSIZE, gpuGbCount);
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gbCount, gpuGbCount, sizeof(int), hipMemcpyDeviceToHost));

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL(hipMalloc((void**)&gpu_psum,HSIZE*sizeof(int)));
        } else {
            alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpu_psum, HSIZE * sizeof(int));
            GPU_MEMPOOL_CHECK(gpu_inner_mp);
        }
        scanImpl(gpu_hashNum,HSIZE,gpu_psum,pp);

        if (!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbCount));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_hashNum));
        }
    }

    if(gbConstant == 1)
        res->tupleNum = 1;
    else
        res->tupleNum = gbCount;

    //printf("[INFO]Number of groupBy results: %ld\n",res->tupleNum);

    char ** gpuResult = NULL;
    char ** result = NULL;

    if(!use_mempool) {
        result = (char **)malloc(sizeof(char*)*res->totalAttr);
        CHECK_POINTER(result);
    } else {
        result = (char **) alloc_mempool(sizeof(char *) * res->totalAttr);
        MEMPOOL_CHECK();
    }
    if(!use_gpu_mempool) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuResult, sizeof(char *)* res->totalAttr));
    } else {
        alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuResult, sizeof(char *) * res->totalAttr);
        GPU_MEMPOOL_CHECK(gpu_inner_mp);
    }

    for(int i=0; i<res->totalAttr;i++){
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&result[i], res->tupleNum * res->attrSize[i]));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(result[i], 0, res->tupleNum * res->attrSize[i]));
        res->content[i] = result[i]; 
        res->dataPos[i] = GPU;
        res->attrTotalSize[i] = res->tupleNum * res->attrSize[i];
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuResult[i], &result[i], sizeof(char *), hipMemcpyHostToDevice));

        if(gb->gbExp[i].func == MIN && res->attrSize[i] == sizeof(int))
            init_int_array<<<grid, block>>>((int *)result[i], res->tupleNum, FLOAT_MAX);
        else if(gb->gbExp[i].func == MAX && res->attrSize[i] == sizeof(int))
            init_int_array<<<grid, block>>>((int *)result[i], res->tupleNum, FLOAT_MIN);
    }


    if(!use_gpu_mempool) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int)*res->totalAttr));
    } else {
        alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbType, sizeof(int) * res->totalAttr);
        GPU_MEMPOOL_CHECK(gpu_inner_mp);
    }
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType, res->attrType, sizeof(int)*res->totalAttr, hipMemcpyHostToDevice));
    if(!use_gpu_mempool) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int)*res->totalAttr));
    } else {
        alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbSize, sizeof(int) * res->totalAttr);
        GPU_MEMPOOL_CHECK(gpu_inner_mp);
    }
    struct groupByExp *gpuGbExp;

    if(!use_gpu_mempool){
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuGbExp, sizeof(struct groupByExp)*res->totalAttr));
    } else {
        alloc_gpu_mempool(&gpu_inner_mp, (char **)&gpuGbExp, sizeof(struct groupByExp) * res->totalAttr);
        GPU_MEMPOOL_CHECK(gpu_inner_mp);
    }
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbExp, gb->gbExp, sizeof(struct groupByExp)*res->totalAttr, hipMemcpyHostToDevice));
    for(int i=0;i<res->totalAttr;i++){
        struct mathExp * tmpMath;
        if(gb->gbExp[i].exp.opNum == 2){
            if(!use_gpu_mempool) {
                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&tmpMath, 2* sizeof(struct mathExp)));
            } else {
                alloc_gpu_mempool(&gpu_inner_mp, (char **)&tmpMath, 2 * sizeof(struct mathExp));
                GPU_MEMPOOL_CHECK(gpu_inner_mp);
            }
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tmpMath,(struct mathExp*)gb->gbExp[i].exp.exp,2*sizeof(struct mathExp), hipMemcpyHostToDevice));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&(gpuGbExp[i].exp.exp), &tmpMath, sizeof(struct mathExp *), hipMemcpyHostToDevice));
        }
    }

    gpuGbColNum = res->totalAttr;

    if(gbConstant !=1){
        agg_cal<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuGbType, gpuGbSize, gpuTupleNum, gpuGbKey, gpu_psum, gpu_groupNum,gpuResult);
        if(!use_gpu_mempool) {
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbKey));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_psum));
            CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_groupNum));
        }
    }else
        agg_cal_cons<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuTupleNum,gpuResult);

    for(int i=0; i<gb->table->totalAttr;i++){
        if(gb->table->dataPos[i]==MEM)
            CUDA_SAFE_CALL_NO_SYNC(hipFree(column[i]));
    }
    if(!use_mempool)
        free(column);
    if(!use_gpu_mempool) {
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuContent));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbExp));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuResult));
    }

    clock_gettime(CLOCK_REALTIME,&end);
    double timeE = (end.tv_sec -  start.tv_sec)* BILLION + end.tv_nsec - start.tv_nsec;
    //printf("GroupBy Time: %lf\n", timeE/(1000*1000));

    return res;
}
